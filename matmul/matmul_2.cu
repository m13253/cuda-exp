#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>

static void print_matrix(const char *name, const float *matrix, int h, int w) {
    int eff_h = std::min(h, 8);
    int eff_w = std::min(w, 8);
    std::printf("%s = [\n", name);
    for(int i = 0; i < eff_h; ++i) {
        for(int j = 0; j < eff_w; ++j) {
            std::printf(" %5g", matrix[i*w+j]);
        }
        if(eff_w == w) {
            std::puts("");
        } else {
            std::puts(" ...");
        }
    }
    if(eff_h == h) {
        std::puts("]\n");
    } else {
        std::puts("... ]\n");
    }
}

__global__ static void matrix_mul(float q[], const float a[], const float b[], int size, int blocks) {
    extern __shared__ float shared[];
    float *tile_a = shared;
    float *tile_b = shared + size;
    int i = threadIdx.x, j = threadIdx.y;
    tile_a[i*size+j] = a[calc_index(blockIdx.x, blockIdx.z, i, j, size, blocks)];
    tile_b[i*size+j] = b[calc_index(blockIdx.z, blockIdx.y, i, j, size, blocks)];
    __syncthreads();
    // STUB
}

static hipError_t report_error(void) {
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        std::fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        std::abort();
    }
    return err;
}

int main(int argc, char *argv[]) {
    int size = 8;
    if(argc >= 2) {
        std::sscanf(argv[1], "%d", &size);
    }
    int block_size = 8;
    if(argc >= 3) {
        std::sscanf(argv[2], "%d", &block_size);
    }
    size = ((size-1)/block_size+1) * block_size;

    float *a = new float[size*size];
    float *b = new float[size*size];
    for(int i = 0; i < size*size; ++i) {
        a[i] = i+1;
    }
    for(int i = 0; i < size*size; ++i) {
        b[i] = i+1;
    }
    print_matrix("a", a, size, size);
    print_matrix("b", b, size, size);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size*size*sizeof *d_a); report_error();
    hipMalloc(&d_b, size*size*sizeof *d_b); report_error();
    hipMalloc(&d_c, size*size*sizeof *d_c); report_error();
    hipMemcpy(d_a, a, size*size*sizeof *a, hipMemcpyHostToDevice); report_error();
    hipMemcpy(d_b, b, size*size*sizeof *b, hipMemcpyHostToDevice); report_error();
    hipMemset(d_c, 0, size*size*sizeof *d_c); report_error();

    matrix_mul<<<dim3(size/block_size, size/block_size, size/block_size), dim3(block_size, block_size), size*sizeof (float)*2>>>(d_c, d_a, d_b, block_size, size/block_size); report_error();

    float *c = new float[size*size];
    hipMemcpy(c, d_c, size*size*sizeof *c, hipMemcpyDeviceToHost); report_error();
    print_matrix("a * b", c, size, size);
    delete[] c;

    hipFree(d_c); report_error();
    hipFree(d_b); report_error();
    hipFree(d_a); report_error();
    delete[] b;
    delete[] a;

    return 0;
}
