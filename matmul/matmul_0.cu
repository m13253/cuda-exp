
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdio>
#include <cstring>

static void print_matrix(const char *name, const float *matrix, int h, int w) {
    int eff_h = std::min(h, 8);
    int eff_w = std::min(w, 8);
    std::printf("%s = [\n", name);
    for(int i = 0; i < eff_h; ++i) {
        for(int j = 0; j < eff_w; ++j) {
            std::printf(" %5g", matrix[i*w+j]);
        }
        if(eff_w == w) {
            std::puts("");
        } else {
            std::puts(" ...");
        }
    }
    if(eff_h == h) {
        std::puts("]\n");
    } else {
        std::puts("... ]\n");
    }
}

__global__ static void matrix_mul(float q[], const float a[], const float b[], int size) {
    int i = threadIdx.x, j = threadIdx.y;
    float s = 0;
    for(int k = 0; k < size; ++k) {
        s += a[i*size+k] * b[k*size+j];
    }
    q[i*size+j] = s;
}

static hipError_t report_error(void) {
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        std::fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        std::abort();
    }
    return err;
}

int main(int argc, char *argv[]) {
    int size = 8;
    if(argc >= 2) {
        std::sscanf(argv[1], "%d", &size);
    }
    bool bench = argc >= 4 && std::strcmp(argv[3], "bench") == 0;

    float *a = new float[size*size];
    float *b = new float[size*size];
    for(int i = 0; i < size*size; ++i) {
        a[i] = i+1;
    }
    for(int i = 0; i < size*size; ++i) {
        b[i] = i+1;
    }
    if(!bench) {
        print_matrix("a", a, size, size);
        print_matrix("b", b, size, size);
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size*size*sizeof *d_a); report_error();
    hipMalloc(&d_b, size*size*sizeof *d_b); report_error();
    hipMalloc(&d_c, size*size*sizeof *d_c); report_error();
    hipMemcpy(d_a, a, size*size*sizeof *a, hipMemcpyHostToDevice); report_error();
    hipMemcpy(d_b, b, size*size*sizeof *b, hipMemcpyHostToDevice); report_error();

    hipEvent_t start, end;
    hipEventCreate(&start); report_error();
    hipEventCreate(&end); report_error();
    hipEventRecord(start); report_error();
    hipEventSynchronize(start); report_error();

    matrix_mul<<<1, dim3(size, size)>>>(d_c, d_a, d_b, size); report_error();

    hipEventRecord(end); report_error();
    hipEventSynchronize(end); report_error();
    float elapsed;
    hipEventElapsedTime(&elapsed, start, end); report_error();
    hipEventDestroy(end); report_error();
    hipEventDestroy(start); report_error();

    if(!bench) {
        float *c = new float[size*size];
        hipMemcpy(c, d_c, size*size*sizeof *c, hipMemcpyDeviceToHost); report_error();
        print_matrix("a * b", c, size, size);
        delete[] c;
        std::printf("%.9g s elapsed\n", elapsed * 0.001f);
    } else {
        std::printf("%.9g", elapsed * 0.001f);
    }

    hipFree(d_c); report_error();
    hipFree(d_b); report_error();
    hipFree(d_a); report_error();
    delete[] b;
    delete[] a;

    return 0;
}
