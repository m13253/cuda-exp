
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdlib>
#include <cstdio>

__global__ static void fill(int a[]) {
    unsigned n = blockIdx.x * blockDim.x + threadIdx.x;
    a[n] = n;
}

__global__ static void scan(int a[], int blk_sum[]) {
    extern __shared__ int slice[];
    unsigned n = threadIdx.x;
    slice[n] = a[blockIdx.x*blockDim.x*2 + n];
    slice[blockDim.x + n] = a[blockIdx.x*blockDim.x*2 + blockDim.x + n];
    __syncthreads();
    unsigned i;
    for(i = 1; i < blockDim.x*2; i *= 2) {
        unsigned from = (n * 2 + 1) * i - 1;
        unsigned to = from + i;
        if(to < blockDim.x*2) {
            slice[to] += slice[from];
        }
        __syncthreads();
    }
    for(i /= 2; i != 0; i /= 2) {
        unsigned from = (n + 1) * i * 2 - 1;
        unsigned to = from + i;
        if(to < blockDim.x*2) {
            slice[to] += slice[from];
        }
        __syncthreads();
    }
    a[blockIdx.x*blockDim.x*2 + n] = slice[n];
    a[blockIdx.x*blockDim.x*2 + blockDim.x + n] = slice[blockDim.x + n];
    if(blk_sum && n == 0) {
        blk_sum[blockIdx.x+1] = slice[blockDim.x*2-1];
    }
}

__global__ static void fix(int a[], int blk_sum[]) {
    a[blockIdx.x*blockDim.x + threadIdx.x] += blk_sum[blockIdx.x/2];
}

static hipError_t report_error(void) {
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        std::fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        std::abort();
    }
    return err;
}

int main(int argc, char *argv[]) {
    unsigned device_id = 0;
    if(argc >= 2 && std::sscanf(argv[1], "-d%u", &device_id) == 1) {
        --argc; ++argv;
    }
    hipSetDevice(device_id); report_error();

    unsigned blk_size = 128;
    if(argc >= 2) {
        std::sscanf(argv[1], "%u", &blk_size);
        blk_size = ((blk_size-1)/4+1)*8;
    }
    unsigned blk_cnt = 2;
    if(argc >= 3) {
        std::sscanf(argv[2], "%u", &blk_cnt);
        blk_cnt = ((blk_cnt-1)/4+1)*4/2;
    }
    unsigned times = 1;
    if(argc >= 4) {
        std::sscanf(argv[3], "%u", &times);
    } else {
        std::fprintf(stderr, "Block configuration: %u x %u (%u)\n", blk_size, blk_cnt, blk_size * blk_cnt);
    }

    int *a;
    hipMalloc(&a, blk_cnt * blk_size * sizeof (int)); report_error();
    fill<<<blk_cnt*2, blk_size/2>>>(a); report_error();

    int *blk_sum;
    hipMalloc(&blk_sum, blk_cnt * sizeof (int)); report_error();
    hipMemset(blk_sum, 0, sizeof (int)); report_error();

    hipEvent_t start, end;
    hipEventCreate(&start); report_error();
    hipEventCreate(&end); report_error();
    hipEventRecord(start); report_error();
    hipEventSynchronize(start); report_error();
    for(unsigned i = 0; i < times; ++i) {
        scan<<<blk_cnt, blk_size/2, blk_size * sizeof (int)>>>(a, blk_sum); report_error();
        scan<<<1, blk_cnt/2, blk_cnt * sizeof (int)>>>(blk_sum, NULL); report_error();
        fix<<<blk_cnt*2, blk_size/2>>>(a, blk_sum); report_error();
    }
    hipEventRecord(end); report_error();
    hipEventSynchronize(end); report_error();
    float elapsed;
    hipEventElapsedTime(&elapsed, start, end); report_error();
    hipEventDestroy(end); report_error();
    hipEventDestroy(start); report_error();

    hipFree(blk_sum); report_error();

    if(argc < 4) {
        size_t n = std::min<size_t>(256, blk_size * blk_cnt);
        int *h_a = new int[n];
        hipMemcpy(h_a, a, n * sizeof (int), hipMemcpyDeviceToHost); report_error();
        std::printf("[%d", h_a[0]);
        for(size_t i = 1; i < n; ++i) {
            std::printf(", %u", h_a[i]);
        }
        std::puts("]");
        delete[] h_a;
    } else {
        std::printf("%.9g", elapsed * 0.001f / times);
    }

    hipFree(a); report_error();

    return 0;
}
