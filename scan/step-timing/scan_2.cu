
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdlib>
#include <cstdio>

__global__ static void fill(int a[]) {
    unsigned n = blockIdx.x * blockDim.x + threadIdx.x;
    a[n] = n;
}

__global__ static void scan(int a[], int blk_sum[]) {
    extern __shared__ int slice[];
    unsigned n = threadIdx.x;
    slice[n] = a[blockIdx.x*blockDim.x*2 + n];
    slice[blockDim.x + n] = a[blockIdx.x*blockDim.x*2 + blockDim.x + n];
    __syncthreads();
    unsigned i;
    for(i = 1; i < blockDim.x*2; i *= 2) {
        unsigned from = (n * 2 + 1) * i - 1;
        unsigned to = from + i;
        if(to < blockDim.x*2) {
            slice[to] += slice[from];
        }
        __syncthreads();
    }
    for(i /= 2; i != 0; i /= 2) {
        unsigned from = (n + 1) * i * 2 - 1;
        unsigned to = from + i;
        if(to < blockDim.x*2) {
            slice[to] += slice[from];
        }
        __syncthreads();
    }
    a[blockIdx.x*blockDim.x*2 + n] = slice[n];
    a[blockIdx.x*blockDim.x*2 + blockDim.x + n] = slice[blockDim.x + n];
    if(blk_sum && n == 0) {
        blk_sum[blockIdx.x+1] = slice[blockDim.x*2-1];
    }
}

__global__ static void fix(int a[], int blk_sum[]) {
    a[blockIdx.x*blockDim.x + threadIdx.x] += blk_sum[blockIdx.x/2];
}

static hipError_t report_error(void) {
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        std::fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        std::abort();
    }
    return err;
}

int main(int argc, char *argv[]) {
    unsigned device_id = 0;
    if(argc >= 2 && std::sscanf(argv[1], "-d%u", &device_id) == 1) {
        --argc; ++argv;
    }
    hipSetDevice(device_id); report_error();

    unsigned blk_size = 128;
    if(argc >= 2) {
        std::sscanf(argv[1], "%u", &blk_size);
        blk_size = ((blk_size-1)/4+1)*8;
    }
    unsigned blk_cnt = 2;
    if(argc >= 3) {
        std::sscanf(argv[2], "%u", &blk_cnt);
        blk_cnt = ((blk_cnt-1)/4+1)*4/2;
    }
    unsigned times = 10;
    if(argc >= 4) {
        std::sscanf(argv[3], "%u", &times);
    }

    int *a;
    hipMalloc(&a, blk_cnt * blk_size * sizeof (int)); report_error();
    fill<<<blk_cnt*2, blk_size/2>>>(a); report_error();

    int *blk_sum;
    hipMalloc(&blk_sum, blk_cnt * sizeof (int)); report_error();
    hipMemset(blk_sum, 0, sizeof (int)); report_error();

    hipEvent_t start, step1, step2, end;
    hipEventCreate(&start); report_error();
    hipEventCreate(&step1); report_error();
    hipEventCreate(&step2); report_error();
    hipEventCreate(&end); report_error();

    hipEventRecord(start); report_error();
    hipEventSynchronize(start); report_error();

    for(unsigned i = 0; i < times; ++i) {
        scan<<<blk_cnt, blk_size/2, blk_size * sizeof (int)>>>(a, blk_sum); report_error();
    }

    hipEventRecord(step1); report_error();
    hipEventSynchronize(step1); report_error();

    for(unsigned i = 0; i < times; ++i) {
        scan<<<1, blk_cnt/2, blk_cnt * sizeof (int)>>>(blk_sum, NULL); report_error();
    }

    hipEventRecord(step2); report_error();
    hipEventSynchronize(step2); report_error();

    for(unsigned i = 0; i < times; ++i) {
        fix<<<blk_cnt*2, blk_size/2>>>(a, blk_sum); report_error();
    }

    hipEventRecord(end); report_error();
    hipEventSynchronize(end); report_error();

    float elapsed01, elapsed12, elapsed23;
    hipEventElapsedTime(&elapsed01, start, step1); report_error();
    hipEventElapsedTime(&elapsed12, step1, step2); report_error();
    hipEventElapsedTime(&elapsed23, step2, end); report_error();

    hipEventDestroy(end); report_error();
    hipEventDestroy(step2); report_error();
    hipEventDestroy(step1); report_error();
    hipEventDestroy(start); report_error();

    hipFree(blk_sum); report_error();
    hipFree(a); report_error();

    std::printf("step 1 scan: %.9g s\n", elapsed01 * 0.001f / times);
    std::printf("step 2 scan: %.9g s\n", elapsed12 * 0.001f / times);
    std::printf("step 3 fix:  %.9g s\n", elapsed23 * 0.001f / times);

    return 0;
}
