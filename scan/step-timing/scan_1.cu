
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdlib>
#include <cstdio>

__global__ static void fill(int a[]) {
    unsigned n = blockIdx.x * blockDim.x + threadIdx.x;
    a[n] = n;
}

__global__ static void scan(int a[], int blk_sum[]) {
    extern __shared__ int slice[];
    unsigned n = threadIdx.x;
    slice[n] = a[blockIdx.x*blockDim.x + n];
    __syncthreads();
    for(int i = 1; i < blockDim.x; i *= 2) {
        int from = n - i;
        int delta = from >= 0 ? slice[from] : 0;
        __syncthreads();
        slice[n] += delta;
        __syncthreads();
    }
    a[blockIdx.x*blockDim.x + n] = slice[n];
    if(blk_sum && n == 0) {
        blk_sum[blockIdx.x+1] = slice[blockDim.x-1];
    }
}

__global__ static void fix(int a[], int blk_sum[]) {
    a[blockIdx.x*blockDim.x + threadIdx.x] += blk_sum[blockIdx.x];
}

static hipError_t report_error(void) {
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        std::fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
        std::abort();
    }
    return err;
}

int main(int argc, char *argv[]) {
    unsigned device_id = 0;
    if(argc >= 2 && std::sscanf(argv[1], "-d%u", &device_id) == 1) {
        --argc; ++argv;
    }
    hipSetDevice(device_id); report_error();

    unsigned blk_size = 64;
    if(argc >= 2) {
        std::sscanf(argv[1], "%u", &blk_size);
        blk_size = ((blk_size-1)/4+1)*4;
    }
    unsigned blk_cnt = 4;
    if(argc >= 3) {
        std::sscanf(argv[2], "%u", &blk_cnt);
    }
    unsigned times = 10;
    if(argc >= 4) {
        std::sscanf(argv[3], "%u", &times);
    }
    printf("Block configuration: %u x %u (%u)\n", blk_size, blk_cnt, blk_size * blk_cnt);
    printf("Test %d times\n", times);

    int *a;
    hipMalloc(&a, blk_cnt * blk_size * sizeof (int)); report_error();
    fill<<<blk_cnt, blk_size>>>(a); report_error();

    int *blk_sum;
    hipMalloc(&blk_sum, blk_cnt * sizeof (int)); report_error();
    hipMemset(blk_sum, 0, sizeof (int)); report_error();

    hipEvent_t start, step1, step2, end;
    hipEventCreate(&start); report_error();
    hipEventCreate(&step1); report_error();
    hipEventCreate(&step2); report_error();
    hipEventCreate(&end); report_error();

    hipEventRecord(start); report_error();
    hipEventSynchronize(start); report_error();

    for(unsigned i = 0; i < times; ++i) {
        scan<<<blk_cnt, blk_size, blk_size * sizeof (int)>>>(a, blk_sum); report_error();
    }

    hipEventRecord(step1); report_error();
    hipEventSynchronize(step1); report_error();

    for(unsigned i = 0; i < times; ++i) {
        scan<<<1, blk_cnt, blk_cnt * sizeof (int)>>>(blk_sum, NULL); report_error();
    }
    hipEventRecord(step2); report_error();
    hipEventSynchronize(step2); report_error();

    for(unsigned i = 0; i < times; ++i) {
        fix<<<blk_cnt, blk_size>>>(a, blk_sum); report_error();
    }

    hipEventRecord(end); report_error();
    hipEventSynchronize(end); report_error();

    float elapsed01, elapsed12, elapsed23;
    hipEventElapsedTime(&elapsed01, start, step1); report_error();
    hipEventElapsedTime(&elapsed12, step1, step2); report_error();
    hipEventElapsedTime(&elapsed23, step2, end); report_error();

    hipEventDestroy(end); report_error();
    hipEventDestroy(step2); report_error();
    hipEventDestroy(step1); report_error();
    hipEventDestroy(start); report_error();

    hipFree(blk_sum); report_error();
    hipFree(a); report_error();

    std::printf("step 1 scan: %.9g s\n", elapsed01 * 0.001f / times);
    std::printf("step 2 scan: %.9g s\n", elapsed12 * 0.001f / times);
    std::printf("step 3 fix:  %.9g s\n", elapsed23 * 0.001f / times);

    return 0;
}
